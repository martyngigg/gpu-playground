#include "hip/hip_runtime.h"
// -*- mode: c++; -*-

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// device variable and kernel
__device__ float d_test = 0.0f;
__global__ void kernel1() { d_test = 1.0; }

int main() {

  // initialise variables
  float h_test = 0.0;
  //hipMemcpyToSymbol(HIP_SYMBOL(d_test),&h_test, sizeof(float));

  // check for error
  hipError_t error = hipGetLastError();
  if(error != hipSuccess)
  {
    // print the CUDA error message and exit
    printf("hipMemset error: %s\n", hipGetErrorString(error));
    exit(-1);
  }

  // invoke kernel
  kernel1 <<<1,1>>> ();

  hipDeviceSynchronize();

  // check for error
  error = hipGetLastError();
  if(error != hipSuccess)
  {
    // print the CUDA error message and exit
    printf("kernel error: %s\n", hipGetErrorString(error));
    exit(-1);
  }

  // Copy device variable to host and print
  hipMemcpyFromSymbol(&h_test,HIP_SYMBOL(d_test),sizeof(float));

  printf("Value of h_test: %f\n:", h_test);

}
